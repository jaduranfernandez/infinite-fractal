#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <cstdlib>
#include "Color.h"
#include "Fractal.h"
#include <thrust/complex.h>

#define THREADS_PER_BLOCK 32

__device__ float linearInterpolationCuda(float value, float a, float b, float c, float d) {
    return c + (((d - c) / (b - a)) * (value - a));
}

__device__ thrust::complex<double> fromPixelToComplex(int x, int y, int width, int height, Fractal fractal) {
    float realValue = linearInterpolationCuda(x, 0, width, fractal.left, fractal.right);
    float imaValue = linearInterpolationCuda(y, 0, height, fractal.up, fractal.down);
    return thrust::complex<double>(realValue, imaValue);
}


__global__ void calculatePixelColor(Fractal fractal, Color* colors, int width, int height) {
    
    int x = threadIdx.x + (blockDim.x * blockIdx.x);
    int y = threadIdx.y + (blockDim.y * blockIdx.y);
    if (y > height - 1 || x > width - 1) return;


    int currentIteration = 0;
    thrust::complex<double> point = fromPixelToComplex(x, y, width, height, fractal);
    thrust::complex<double> previousCoordinate = point;
    thrust::complex<double> nextCoordinate = point*point + point;
    float magDifference = fractal.tolerance + 1;
    while (currentIteration < fractal.maxIterations && magDifference > fractal.tolerance) {
        previousCoordinate = nextCoordinate;
        nextCoordinate = nextCoordinate * nextCoordinate + point;
        nextCoordinate = nextCoordinate * nextCoordinate + point;
        magDifference = abs(nextCoordinate - previousCoordinate);
        currentIteration++;
    }
    float intensity = (currentIteration - 1) / (float)fractal.maxIterations;
    //  Color lerping
    colors[y * width + x].r = fractal.color.r + (fractal.altColor.r - fractal.color.r) * intensity;
    colors[y * width + x].g = fractal.color.g + (fractal.altColor.g - fractal.color.g) * intensity;
    colors[y * width + x].b = fractal.color.b + (fractal.altColor.b - fractal.color.b) * intensity;


    /*colors[y * width + x].r = fractal.color.r * intensity;
    colors[y * width + x].g = fractal.color.g * intensity;
    colors[y * width + x].b = fractal.color.b * intensity;*/
}

void parallelFractal(Fractal fractal, Color* h_Colors, int width, int height) {
    size_t numbytes = width * height * sizeof(Color);
    
    Color* d_Colors;


    hipMalloc((void**)&d_Colors, numbytes);

    hipMemset(d_Colors, 0, numbytes);

    hipMemcpy(d_Colors, h_Colors, numbytes, hipMemcpyHostToDevice);

    const dim3 blockSize(THREADS_PER_BLOCK, THREADS_PER_BLOCK);
    const dim3 gridSize(ceil(width / blockSize.x + 1), ceil(height / blockSize.y + 1));

    calculatePixelColor << < gridSize, blockSize >> > (fractal, d_Colors, width, height);
    hipMemcpy(h_Colors, d_Colors, numbytes, hipMemcpyDeviceToHost);
    

    //Free memory
    hipFree(d_Colors);
}



