#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <cstdlib>
#include "Color.h"
#include "Fractal.h"
#include <thrust/complex.h>

#define THREADS_PER_BLOCK 8


__global__ void colorTest(Fractal fractal, Color* colors, int width, int height) {
    
    int col = threadIdx.x + (blockDim.x * blockIdx.x);
    int row = threadIdx.y + (blockDim.y * blockIdx.y);
    if (row > height - 1 || col > width - 1) return;


    //printf("(%i,%i) \n", row, col);
    //colors[row * width + col] = Color(0, 0, 255);

    int currentIteration = 0;
    //  (y, 0, windowHeight, up, down)
    float real  = fractal.left + (((fractal.right - fractal.left) / (width)) * (col));
    float image = fractal.up + (((fractal.down - fractal.up) / (height)) * (row));
    thrust::complex<double> point(real,image);
    thrust::complex<double> previousCoordinate = point;
    thrust::complex<double> nextCoordinate = point*point + point;
    float magDifference = fractal.tolerance + 1;
    while (currentIteration < fractal.maxIterations && magDifference > fractal.tolerance) {
        previousCoordinate = nextCoordinate;
        nextCoordinate = nextCoordinate * nextCoordinate + point;
        nextCoordinate = nextCoordinate * nextCoordinate + point;
        magDifference = abs(nextCoordinate - previousCoordinate);
        currentIteration++;
    }
    float intensity = currentIteration / (float)fractal.maxIterations;

    colors[row * width + col].r = fractal.color.r * intensity;
    colors[row * width + col].g = fractal.color.g * intensity;
    colors[row * width + col].b = fractal.color.b * intensity;
}

void parallelFractal(Fractal fractal, Color* h_Colors, int width, int height) {
    size_t numbytes = width * height * sizeof(Color);

    
    Color* d_Colors;


    hipMalloc((void**)&d_Colors, numbytes);

    hipMemset(d_Colors, 0, numbytes);

    hipMemcpy(d_Colors, h_Colors, numbytes, hipMemcpyHostToDevice);

    //Call kernel function

    const dim3 blockSize(THREADS_PER_BLOCK, THREADS_PER_BLOCK);
    const dim3 gridSize(ceil(width / blockSize.x + 1), ceil(height / blockSize.y + 1));

    colorTest << < gridSize, blockSize >> > (fractal, d_Colors, width, height);
    hipMemcpy(h_Colors, d_Colors, numbytes, hipMemcpyDeviceToHost);
    //Print result
    


    //Free memory
    hipFree(d_Colors);
}


void parallelFractalTest(Color* colors, int width, int height) {

    for (int y = 0; y < height; y++) {
        for (int x = 0; x < width; x++) {
            colors[y * width + x].r = 255;
        }
    }


}

